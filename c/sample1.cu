
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 256

// define kernel functions
__global__ void arrayadd(float *fOut, float *fInA, float *fInB){
  int id = threadIdx.x + blockIdx.x  * blockDim.x;
  fOut[id] = fInA[id] +fInB[id];
}

int main(int argc, char**argv){
  int i;
  const int ishow=16;
  printf("GPU:\n");
  srand(0);

  hipSetDevice(0);
  // variables in host
  float *h_InA, *h_InB, *h_Out;
  h_InA = (float*)malloc(sizeof(float)*SIZE);
  h_InB = (float*)malloc(sizeof(float)*SIZE);
  h_Out = (float*)malloc(sizeof(float)*SIZE);

  // initialize
  for(i=0;i<SIZE;i++) h_InA[i] = (float)(rand()%10)/10.0f;
  for(i=0;i<SIZE;i++) h_InB[i] = (float)(rand()%10)/10.0f; 

  // confirm
  printf("InA: "); for(i=0;i<ishow;i++) printf(" %.2f",h_InA[i]); printf("\n");
  printf("InB: "); for(i=0;i<ishow;i++) printf(" %.2f",h_InB[i]); printf("\n");

  // variables in device
  float *d_InA, *d_InB, *d_Out;
  hipMalloc((void**)&d_InA, sizeof(float)*SIZE);
  hipMalloc((void**)&d_InB, sizeof(float)*SIZE);
  hipMalloc((void**)&d_Out, sizeof(float)*SIZE);
  
  // transfer from host to device
  hipMemcpy(d_InA, h_InA, sizeof(float)*SIZE, hipMemcpyHostToDevice);
  hipMemcpy(d_InB, h_InB, sizeof(float)*SIZE, hipMemcpyHostToDevice);
  hipMemcpy(d_Out, h_Out, sizeof(float)*SIZE, hipMemcpyHostToDevice);

  // call kernel functions, specify grid and block as <<< grid, block >>>
  // 1D decomposition. SIZE = grid1d * block1d
  const int block1d=16;// 16 threads are used.
  const int  grid1d=SIZE/block1d; //grid size is determined by total size and the thread number
  arrayadd<<< grid1d,block1d >>> (d_Out,d_InA, d_InB);    
  hipDeviceSynchronize();
  
  // transfer from device to host
  hipMemcpy(h_Out, d_Out, sizeof(float)*SIZE, hipMemcpyDeviceToHost);
 
  // confirm
  printf("Out: "); for(i=0;i<ishow;i++) printf(" %.2f",h_Out[i]); printf("\n");
  
  return 0;
}

